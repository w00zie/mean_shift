#include "hip/hip_runtime.h"
#include <cassert>
#include "../constants_test_2d.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "../../../utils.h"

// Dataset
constexpr auto N = mean_shift::cuda::test_2d::case_500::N;
constexpr auto D = mean_shift::cuda::test_2d::case_500::D;
constexpr auto M = mean_shift::cuda::test_2d::M;
const auto PATH_TO_DATA = mean_shift::cuda::test_2d::case_500::PATH_TO_DATA; 
const auto PATH_TO_CENTROIDS = mean_shift::cuda::test_2d::case_500::PATH_TO_CENTROIDS;
// Hyperparams
constexpr auto RADIUS = mean_shift::cuda::test_2d::case_500::RADIUS;
constexpr auto MIN_DISTANCE = mean_shift::cuda::test_2d::case_500::MIN_DISTANCE;
constexpr auto NUM_ITER = mean_shift::cuda::test_2d::NUM_ITER;
constexpr auto DBL_SIGMA_SQ = mean_shift::cuda::test_2d::case_500::DBL_SIGMA_SQ;
constexpr auto DIST_TO_REAL = mean_shift::cuda::test_2d::DIST_TO_REAL;
// Device
constexpr auto THREADS = mean_shift::cuda::test_2d::THREADS;
constexpr auto BLOCKS = mean_shift::cuda::test_2d::case_500::BLOCKS;

__global__ void mean_shift_naive(float *data, float *data_next) {
    size_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (tid < N) {
        size_t row = tid * D;
        float new_position[D] = {0.};
        float tot_weight = 0.;
        for (size_t i = 0; i < N; ++i) {
            size_t row_n = i * D;
            float sq_dist = 0.;
            for (size_t j = 0; j < D; ++j) {
                sq_dist += (data[row + j] - data[row_n + j]) * (data[row + j] - data[row_n + j]);
            }
            if (sq_dist <= RADIUS) {
                float weight = expf(-sq_dist / DBL_SIGMA_SQ);
                for (size_t j = 0; j < D; ++j) {
                    new_position[j] += weight * data[row_n + j];
                }
                tot_weight += weight;
            }
        }
        for (size_t j = 0; j < D; ++j) {
            data_next[row + j] = new_position[j] / tot_weight;
        }
    }
    return;
}

int main() {
    mean_shift::cuda::utils::print_info(PATH_TO_DATA, N, D, BLOCKS, THREADS);
    // Load data
    std::array<float, N * D> data = mean_shift::cuda::utils::load_csv<N, D>(PATH_TO_DATA, ',');
    std::array<float, N * D> data_next {};
    float *dev_data;
    float *dev_data_next;
    // Allocate GPU memory
    size_t data_bytes = N * D * sizeof(float);
    hipMalloc(&dev_data, data_bytes);
    hipMalloc(&dev_data_next, data_bytes);
    // Copy to GPU memory
    hipMemcpy(dev_data, data.data(), data_bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_data_next, data_next.data(), data_bytes, hipMemcpyHostToDevice);
    // Run mean shift clustering
    for (size_t i = 0; i < NUM_ITER; ++i) {
        mean_shift_naive<<<BLOCKS, THREADS>>>(dev_data, dev_data_next);
        hipDeviceSynchronize();
        mean_shift::cuda::utils::swap(dev_data, dev_data_next);
    }
    hipMemcpy(data.data(), dev_data, data_bytes, hipMemcpyDeviceToHost);
    // Copy from GPU and de-allocate
    hipFree(dev_data);
    hipFree(dev_data_next);
    // Reduce data to cluster centers
    const auto centroids = mean_shift::cuda::utils::reduce_to_centroids<N, D>(data, MIN_DISTANCE);
    // Check if correct number
    assert(centroids.size() == M);
    // Check if these centroids are sufficiently close to real ones
    const std::array<float, M * D> real = mean_shift::cuda::utils::load_csv<M, D>(PATH_TO_CENTROIDS, ',');
    const bool are_close = mean_shift::cuda::utils::are_close_to_real<M, D>(centroids, real, DIST_TO_REAL);
    assert(are_close);
    std::cout << "\nSUCCESS!\n";
    return 0;

}